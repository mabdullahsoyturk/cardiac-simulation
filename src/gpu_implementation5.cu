#include <math.h>
#include <stdlib.h>
#include <string.h>

#include <iomanip>

#include "kernels.h"
#include "utils.h"

int main(int argc, char **argv) {
  // E is the "Excitation" variable, R is the "Recovery" variable
  // E_prev is the Excitation variable for the previous timestep, and is used in time integration
  double *E, *R, *E_prev;
  double *d_E, *d_R, *d_E_prev;

  double T = 1000.0;
  int m = 200, n = 200;
  int plot_freq = 0;
  int bx = 1, by = 1;
  int kernel = 1;

  cmdLine(argc, argv, T, n, bx, by, plot_freq, kernel);
  m = n;

  CUDA_CALL(hipHostMalloc(&E, sizeof(double) * (n + 2) * (m + 2)));
  CUDA_CALL(hipHostMalloc(&E_prev, sizeof(double) * (n + 2) * (m + 2)));
  CUDA_CALL(hipHostMalloc(&R, sizeof(double) * (n + 2) * (m + 2)));

  CUDA_CALL(hipMalloc(&d_E, sizeof(double) * (n + 2) * (m + 2)));
  CUDA_CALL(hipMalloc(&d_R, sizeof(double) * (n + 2) * (m + 2)));
  CUDA_CALL(hipMalloc(&d_E_prev, sizeof(double) * (n + 2) * (m + 2)));

  initSolutionArrays(E, R, E_prev, m, n);

  double dx = 1.0 / n;

  // For time integration, these values shouldn't change
  double rp = kk * (b + 1) * (b + 1) / 4;
  double dte = (dx * dx) / (d * 4 + ((dx * dx)) * (rp + kk));
  double dtr = 1 / (epsilon + ((M1 / M2) * rp));
  double dt = (dte < dtr) ? 0.95 * dte : 0.95 * dtr;
  double alpha = d * dt / (dx * dx);

  // dumpPrerunInfo(n, T, dt, bx, by, kernel);

  // Kernel config
  int THREADS = 32;

  int BLOCKS = n / THREADS;
  std::cerr << "threads(" << THREADS << "," << THREADS << ")" << std::endl;
  std::cerr << "blocks(" << BLOCKS << "," << BLOCKS << ")" << std::endl;

  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);


  int num_iterations = (int)(T / dt) + 1;
  std::cerr << "T: " << T << ", dt: " << dt << ", x: " << ((int)(T / dt) + 1) << std::endl;

  hostToDeviceCopy(d_E, d_R, d_E_prev, E, R, E_prev, m + 2, n + 2);

  void *kernelArgs[] = {(void *)&d_E, (void *)&d_E_prev,
                        (void *)&d_R, (void *)&alpha,
                        (void *)&n,   (void *)&m,
                        (void *)&kk,  (void *)&dt,
                        (void *)&a,   (void *)&epsilon,
                        (void *)&M1,  (void *)&M2,
                        (void *)&b,   (void *)&num_iterations};

  double t0 = getTime();  // Start the timer
  CUDA_CALL(hipLaunchCooperativeKernel((void *)kernel5, blocks, threads, kernelArgs, 0, 0));
  hipDeviceSynchronize();
  double time_elapsed = getTime() - t0;
  deviceToHostCopy(E, R, E_prev, d_E, d_R, d_E_prev, m + 2, n + 2);

  // dumpit(E_prev, m);

  dumpPostrunInfo(num_iterations, time_elapsed, m, n, E_prev);

  if (plot_freq) {
    cout << "\n\nEnter any input to close the program and the plot..." << endl;
    getchar();
  }

  hipHostFree(E);
  hipHostFree(E_prev);
  hipHostFree(R);
  hipFree(d_E);
  hipFree(d_R);
  hipFree(d_E_prev);

  return 0;
}
