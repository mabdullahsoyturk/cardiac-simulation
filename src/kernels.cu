#include "hip/hip_runtime.h"
#include "kernels.h"

__global__ void kernel1_pde(double* E, double* E_prev, double* R, const double alpha, const int n, const int m,
                            const double kk, const double dt, const double a, const double epsilon, const double M1,
                            const double M2, const double b) {
  int row_index = blockIdx.y * blockDim.y + threadIdx.y + 1;
  int column_index = blockIdx.x * blockDim.x + threadIdx.x + 1;

  int row = row_index * (n + 2);
  const int last_row = (m + 1) * (n + 2);
  const int last_row_inner = (m - 1) * (n + 2);

  if (row_index <= m && threadIdx.x == 0) {
    E_prev[row] = E_prev[row + 2];
    E_prev[row + n + 1] = E_prev[row + n - 1];
  }

  if (column_index <= n && threadIdx.y == 0) {
    E_prev[column_index] = E_prev[2 * (m + 2) + column_index];
    E_prev[last_row + column_index] = E_prev[last_row_inner + column_index];
  }

  __syncthreads();

  if (column_index <= n && row_index <= m) {
    int flattened_index = row + column_index;
    E[flattened_index] =
        E_prev[flattened_index] +
        alpha * (E_prev[flattened_index + 1] + E_prev[flattened_index - 1] -
                 4 * E_prev[flattened_index] + E_prev[(row_index + 1) * (n + 2) + column_index] +
                 E_prev[(row_index - 1) * (n + 2) + column_index]);
  }
}

__global__ void kernel1_ode(double* E, double* E_prev, double* R, const double alpha, const int n, const int m,
                            const double kk, const double dt, const double a, const double epsilon, const double M1,
                            const double M2, const double b) {
  int row_index = blockIdx.y * blockDim.y + threadIdx.y + 1;
  int column_index = blockIdx.x * blockDim.x + threadIdx.x + 1;

  int row = row_index * (n + 2);

  if (column_index <= n && row_index <= m) {
    int flattened_index = row + column_index;

    E[flattened_index] =
        E[flattened_index] -
        dt * (kk * E[flattened_index] * (E[flattened_index] - a) *
                  (E[flattened_index] - 1) +
              E[flattened_index] * R[flattened_index]);

    R[flattened_index] =
        R[flattened_index] +
        dt * (epsilon + M1 * R[flattened_index] / (E[flattened_index] + M2)) *
            (-R[flattened_index] -
             kk * E[flattened_index] * (E[flattened_index] - b - 1));
  }
}

__global__ void kernel2(double* E, double* E_prev, double* R, const double alpha, const int n, const int m,
                        const double kk, const double dt, const double a, const double epsilon, const double M1,
                        const double M2, const double b) {
  int row_index = blockIdx.y * blockDim.y + threadIdx.y + 1;
  int column_index = blockIdx.x * blockDim.x + threadIdx.x + 1;

  int row = row_index * (n + 2);
  const int last_row = (m + 1) * (n + 2);
  const int last_row_inner = (m - 1) * (n + 2);

  if (row_index <= m && threadIdx.x == 0) {
    E_prev[row] = E_prev[row + 2];
    E_prev[row + n + 1] = E_prev[row + n - 1];
  }

  if (column_index >= 1 && column_index <= n && threadIdx.y == 0) {
    E_prev[column_index] = E_prev[2 * (m + 2) + column_index];
    E_prev[last_row + column_index] = E_prev[last_row_inner + column_index];
  }

  __syncthreads();

  if (column_index <= n && row_index <= m) {
    int flattened_index = row + column_index;
    E[flattened_index] =
        E_prev[flattened_index] +
        alpha * (E_prev[flattened_index + 1] + E_prev[flattened_index - 1] -
                 4 * E_prev[flattened_index] + E_prev[(row_index + 1) * (n + 2) + column_index] +
                 E_prev[(row_index - 1) * (n + 2) + column_index]);

    E[flattened_index] =
        E[flattened_index] -
        dt * (kk * E[flattened_index] * (E[flattened_index] - a) *
                  (E[flattened_index] - 1) +
              E[flattened_index] * R[flattened_index]);

    R[flattened_index] =
        R[flattened_index] +
        dt * (epsilon + M1 * R[flattened_index] / (E[flattened_index] + M2)) *
            (-R[flattened_index] -
             kk * E[flattened_index] * (E[flattened_index] - b - 1));
  }
}

__global__ void kernel3(double* E, double* E_prev, double* R, const double alpha, const int n, const int m,
                        const double kk, const double dt, const double a, const double epsilon, const double M1,
                        const double M2, const double b) {
  int row_index = blockIdx.y * blockDim.y + threadIdx.y + 1;
  int column_index = blockIdx.x * blockDim.x + threadIdx.x + 1;

  int row = row_index * (n + 2);
  const int last_row = (m + 1) * (n + 2);
  const int last_row_inner = (m - 1) * (n + 2);

  if (row_index <= m && threadIdx.x == 0) {
    E_prev[row] = E_prev[row + 2];
    E_prev[row + n + 1] = E_prev[row + n - 1];
  }

  if (column_index <= n && threadIdx.y == 0) {
    E_prev[column_index] = E_prev[2 * (m + 2) + column_index];
    E_prev[last_row + column_index] = E_prev[last_row_inner + column_index];
  }

  __syncthreads();

  if (column_index <= n && row_index <= m) {
    int flattened_index = row + column_index;
    E[flattened_index] =
        E_prev[flattened_index] +
        alpha * (E_prev[flattened_index + 1] + E_prev[flattened_index - 1] -
                 4 * E_prev[flattened_index] + E_prev[(row_index + 1) * (n + 2) + column_index] +
                 E_prev[(row_index - 1) * (n + 2) + column_index]);

    double temp = E[flattened_index];
    double temp2 = R[flattened_index];

    E[flattened_index] = temp - dt * (kk * temp * (temp - a) * (temp - 1) + temp * temp2);
    temp = E[flattened_index];
    R[flattened_index] =
        temp2 + dt * (epsilon + M1 * temp2 / (temp + M2)) * (-temp2 - kk * temp * (temp - b - 1));
  }
}

__global__ void kernel4(double* E, double* E_prev, double* R, const double alpha, const int n, const int m,
                        const double kk, const double dt, const double a, const double epsilon, const double M1,
                        const double M2, const double b) {
  int row_index = blockIdx.y * blockDim.y + threadIdx.y + 1;
  int column_index = blockIdx.x * blockDim.x + threadIdx.x + 1;

  int row = row_index * (n + 2);
  const int last_row = (m + 1) * (n + 2);
  const int last_row_inner = (m - 1) * (n + 2);

  __shared__ double cache[34][34];

  if (row_index <= m && threadIdx.x == 0) {
    double temp = E_prev[row + 2];
    double temp2 = E_prev[row + n - 1];

    E_prev[row] = temp;
    E_prev[row + n + 1] = temp2;
  }

  if (column_index <= n && threadIdx.y == 0) {
    double temp = E_prev[2 * (m + 2) + column_index];
    double temp2 = E_prev[last_row_inner + column_index];

    E_prev[column_index] = temp;
    E_prev[last_row + column_index] = temp2;
  }

  if (column_index <= n && row_index <= m) {
    int flattened_index = row + column_index;
    
    cache[threadIdx.y + 1][threadIdx.x + 1] = E_prev[flattened_index];

    if (threadIdx.x == 0) {
      cache[threadIdx.y + 1][0] = E_prev[flattened_index - 1];
      cache[threadIdx.y + 1][33] = E_prev[flattened_index + 32];
    }

    if (threadIdx.y == 0) {
      cache[0][threadIdx.x + 1] = E_prev[(row_index - 1) * (n + 2) + column_index];
      cache[33][threadIdx.x + 1] = E_prev[(row_index + 32) * (n + 2) + column_index];
    }
    __syncthreads();

    E[flattened_index] =
        cache[threadIdx.y + 1][threadIdx.x + 1] +
        alpha * (cache[threadIdx.y + 1][threadIdx.x + 2] + cache[threadIdx.y + 1][threadIdx.x] -
                 4 * cache[threadIdx.y + 1][threadIdx.x + 1] + cache[threadIdx.y + 2][threadIdx.x + 1] +
                 cache[threadIdx.y][threadIdx.x + 1]);

    double temp = E[flattened_index];
    double temp2 = R[flattened_index];

    E[flattened_index] = temp - dt * (kk * temp * (temp - a) * (temp - 1) + temp * temp2);
    temp = E[flattened_index];
    R[flattened_index] =
        temp2 + dt * (epsilon + M1 * temp2 / (temp + M2)) * (-temp2 - kk * temp * (temp - b - 1));
  }
}

__global__ void kernel5(double* E, double* E_prev, double* R, const double alpha, const int n, const int m,
                        const double kk, const double dt, const double a, const double epsilon, const double M1,
                        const double M2, const double b, const int num_iterations) {
  int row_index = blockIdx.y * blockDim.y + threadIdx.y + 1;
  int column_index = blockIdx.x * blockDim.x + threadIdx.x + 1;

  int row = row_index * (n + 2);
  const int last_row = (m + 1) * (n + 2);
  const int last_row_inner = (m - 1) * (n + 2);

  cg::thread_block cta = cg::this_thread_block();
  cg::grid_group grid = cg::this_grid();

  int iteration = 0;

  while (iteration < num_iterations) {
    if (row_index <= m && threadIdx.x == 0) {
      E_prev[row] = E_prev[row + 2];
      E_prev[row + n + 1] = E_prev[row + n - 1];
    }

    if (column_index <= n && threadIdx.y == 0) {
      E_prev[column_index] = E_prev[2 * (m + 2) + column_index];
      E_prev[last_row + column_index] = E_prev[last_row_inner + column_index];
    }

    __syncthreads();

    if (column_index <= n && row_index <= m) {
      int flattened_index = row + column_index;
      E[flattened_index] =
          E_prev[flattened_index] +
          alpha * (E_prev[flattened_index + 1] + E_prev[flattened_index - 1] -
                   4 * E_prev[flattened_index] + E_prev[(row_index + 1) * (n + 2) + column_index] +
                   E_prev[(row_index - 1) * (n + 2) + column_index]);

      double temp = E[flattened_index];
      double temp2 = R[flattened_index];

      E[flattened_index] = temp - dt * (kk * temp * (temp - a) * (temp - 1) + temp * temp2);
      temp = E[flattened_index];
      R[flattened_index] =
          temp2 + dt * (epsilon + M1 * temp2 / (temp + M2)) * (-temp2 - kk * temp * (temp - b - 1));
    }

    double* temp_pointer = E;
    E = E_prev;
    E_prev = temp_pointer;

    iteration++;
    grid.sync();
  }

  /*if(blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0) {
    for(int i = 0; i < m+2; i++) {
      for(int j = 0; j < n+2; j++) {
        //printf("Iteration: %d, E[%d][%d]=%f, E_prev[%d][%d]=%f\n", iteration, i, j, E[i * (n+2) + j], i, j, E_prev[i *
  (n+2) + j]); printf("E[%d][%d]:%f\n", i, j, E[i * (n+2) + j]);
      }
    }
  }*/
}